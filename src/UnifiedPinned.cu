/*
Compile:
    nvcc benchmark_pinned_vs_unified.cu -o bench
Run:
    ./bench

This code times a simple vector add with both pinned (page-locked) host memory
and unified memory. For pinned memory, we explicitly do host->device and device->host
copies. For unified memory, we directly access the same pointers from CPU and GPU.

If you don't see a meaningful difference with n=1<<25 (which is ~128MB of floats),
try increasing n to 1<<26 or 1<<27.
*/

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void vecAdd(const float* A, const float* B, float* C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}

float runPinnedVectorAdd(int n) {
    // Timers
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t size = n * sizeof(float);

    // Allocate host pinned memory
    float *h_A, *h_B, *h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    // Init
    for(int i = 0; i < n; i++){
        h_A[i] = 1.f; 
        h_B[i] = 2.f;
    }

    // Device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Start timer
    hipEventRecord(start);

    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize  = (n + blockSize - 1) / blockSize;
    vecAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);

    // Copy back
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Stop timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0.f;
    hipEventElapsedTime(&ms, start, stop);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return ms;
}

float runUnifiedVectorAdd(int n) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t size = n * sizeof(float);

    // Unified memory
    float *A, *B, *C;
    hipMallocManaged(&A, size);
    hipMallocManaged(&B, size);
    hipMallocManaged(&C, size);

    // Init
    for(int i = 0; i < n; i++){
        A[i] = 1.f;
        B[i] = 2.f;
    }

    // Start timer
    hipEventRecord(start);

    // Kernel launch
    int blockSize = 256;
    int gridSize  = (n + blockSize - 1) / blockSize;
    vecAdd<<<gridSize, blockSize>>>(A, B, C, n);

    // Sync
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0.f;
    hipEventElapsedTime(&ms, start, stop);

    // Access results on CPU (already available)
    // (Optionally verify one element)
    // printf("C[0] = %f\n", C[0]);

    // Cleanup
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return ms;
}

int main() {
    // For a real difference, try 1<<25 (~128MB) or higher.
    const int n = 1 << 25;

    float timePinned   = runPinnedVectorAdd(n);
    float timeUnified  = runUnifiedVectorAdd(n);

    printf("Pinned   : %f ms\n", timePinned);
    printf("Unified  : %f ms\n", timeUnified);
    return 0;
}
